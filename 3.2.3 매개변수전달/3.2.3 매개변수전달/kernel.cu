#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include <iostream>

static void HandleError(hipError_t err,
    const char* file,
    int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))

//여기에 커널 작성
__global__ void add(int a,int b, int *c)
{
    *c = a + b;
}

int main()
{
    int c;
    int* dev_c;
    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));

    add << <1, 1 >> > (2, 7, dev_c);

    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
    printf("2+7=%d\n", c);
    hipFree(dev_c);

    return 0;
}