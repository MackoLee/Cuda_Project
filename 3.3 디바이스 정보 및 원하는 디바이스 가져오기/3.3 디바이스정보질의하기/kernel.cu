#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include <iostream>

using namespace std;

static void HandleError(hipError_t err,
    const char* file,
    int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))

//여기에 커널 작성
__global__ void add(int a, int b, int* c)
{
}

int main()
{
    hipDeviceProp_t prop;
    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count)); //그래픽 카드의 갯수를 가져올 수 있다. 나는 지금 한개니깐 0 번째만 가져와도 상관은 없다.
    for (int i = 0; i < count; i++) {
        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        printf(" --- General Information for device %d ---\n", i);
        printf("Name: %s\n", prop.name);
        printf("Compute capability: %d. %d\n", prop.major, prop.minor);
        printf("Clock rate: %d\n", prop.clockRate);
        printf("Device copy overlap: %s\n", (prop.deviceOverlap?"Enabled":"Disabled"));
        printf("Kernel execition timeout: %s\n", (prop.kernelExecTimeoutEnabled ? "Enabled" : "Disabled"));

        puts("");
        printf(" --- Memory Information for device %d ---\n", i);
        printf("Total global mem: %llu\n", prop.totalGlobalMem); //책과의 차이점.. 아주많은 발달로 엄청난 크기의 메모리를 가지고 있어서 ld 가 아닌 llu-> unsigned long long 을 써야한다.
        printf("Total constant mem: %llu\n", prop.totalConstMem);
        printf("Max mem pitch: %llu\n", prop.memPitch);
        printf("Texture Alignment: %llu\n", prop.textureAlignment);
       
        puts("");
        printf(" --- MP Information for device %d ---\n", i);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp: %llu\n", prop.sharedMemPerBlock);
        printf("Registers per mp: %d\n", prop.regsPerBlock);
        printf("Threads in warp: %d\n", prop.warpSize);
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimension: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        puts("");
    }

    // 아래 방법을 이용하여 원하는 버전의 GPU를 고를수 있다.
    // 내컴퓨터는 하나밖에 안가지고 있으므로 멀 찾아도 0번째가 나온다.
    int dev;
    HANDLE_ERROR(hipGetDevice(&dev));
    printf("ID off current CUDA device: %d\n", dev);
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 7;
    prop.minor = 5;
    HANDLE_ERROR(hipChooseDevice(&dev, &prop));
    printf("ID of CUDA device closest to revision 1.3: %d\n", dev);
    HANDLE_ERROR(hipSetDevice(dev));

    return 0;
}